#include "hip/hip_runtime.h"
#include "JacobiRWD.cuh"

float* initMat2D(bool init, bool host){
	return initMat2DHelper(init, host, N);
}

float* initMat2DHelper(bool init, bool host, int size){
	float* m;
	size_t sizeMat = sizeof(float) * size * size;
	if(host){
		m = (float*) malloc(sizeMat);
	}
	else{
		hipMalloc((void **) &m, sizeMat);
	}
	if(!init){
		return m;
	}
	for(int i = 0; i < size; i++){
		for(int j = 0; j < size; j++){
			m[(i * size) + j] = 5;
		}
	}
	return m;
}

float* initMat1D(bool init, bool host){
	return initMat1DHelper(init, host, N);
}

float* initMat1DHelper(bool init, bool host, int size){
	float* m;
	size_t sizeMat = sizeof(float) * size;
	if(host){
		m = (float*) malloc(sizeMat);
	}
	else{
		hipMalloc((void **) &m, sizeMat);
	}
	if(!init){
		return m;
	}
	for(int i = 0; i < size; i++){
		m[i] = 5;
	}
	return m;
}

void printMat1D(float* mat){
	printMat1DHelper(mat, N);
}

void printMat1DHelper(float* mat, int size){
	for(int i = 0; i < size; i++){
		printf("%f\n", mat[i]);
	}
}

void printMat2D(float* mat){
	printMat2DHelper(mat, N);
}

void printMat2DHelper(float* mat, int size){
	for(int i = 0; i < size; i++){
		for(int j = 0; j < size; j++){
			printf("%f ", mat[(i * size) + j]);
		}
		printf("\n");
	}
}

__global__ void multiplyMats2D(float* a, float* b, float* c){
	multiplyMats2DHelper(a, b, c, N, NT, NB);
}

__global__ void multiplyMats2DTB(float* a, float* b, float* c, int lN, int lNT, int lNB){
	multiplyMats2DHelper(a, b, c, lN, lNT, lNB);
}

__device__ void multiplyMats2DHelper(float* a, float* b, float* c, int lN, int lNT, int lNB){
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int bx = blockIdx.x;
	int by = blockIdx.y;
	for(int i = 0; i < lNB; i++){
		for(int j = 0; j < lNB; j++){
			float sum = 0;
			for(int k = 0; k < lN; k++){
				float tmp1 = a[(by * lN * lNT * lNB) + (ty * lN * lNB) + (j * lN) + k];
				float tmp2 = b[(bx * lNT * lNB) + (tx * lNB) + (k * lN) + i];
				//printf("a: %d\n", (by * N * NT * NB) + (ty * N * NB) + (j * N) + k);
				//printf("b: %d\n", (bx * NT * NB) + (tx * NB) + (k * N) + i);
				sum += tmp1 * tmp2;
			}
			c[(by * lN * lNT * lNB) + (bx * lNT * lNB) + (ty * lN * lNB) + (j * lN) + i + (tx * lNB)] = sum;
			//printf("c index %d\n", (by * N * NT * NB) + (bx * NT * NB) + (ty * N * NB) + (j * N) + i + (tx * NB));
		}
	}
}
__global__ void multiplyMats2D1D(float* a, float* b, float* c){
	multiplyMats2D1DHelper(a, b, c, N, NT, NB);
}

__global__ void multiplyMats2D1DTB(float* a, float* b, float* c, int lN, int lNT, int lNB){
	multiplyMats2D1DHelper(a, b, c, lN, lNT, lNB);
}

__device__ void multiplyMats2D1DHelper(float* a, float* b, float* c, int lN, int lNT, int lNB){
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int bx = blockIdx.x;
	int by = blockIdx.y;
	for(int i = 0; i < lNB; i++){
		float sum = 0;
		for(int k = 0; k < lN; k++){
			float tmp1 = a[(by * lN * lNT * lNB) + (tx * lN * lNB) + k];
			float tmp2 = b[(by * lN * lNT * lNB) + (ty * lN * lNB) + k];
			sum += tmp1 * tmp2;
		}
		c[(by * lN * lNT * lNB) + (bx * lNT * lNB) + (ty * lN * lNB) + i + (tx * lNB)] = sum;
	}
}

void addMats2D(float* a, float* b, float* c){
	for(int i = 0; i < N * N; i++){
		c[i] = a[i] + b[i];
	}
}

void subMats2D(float* a, float* b, float* c){
	for(int i = 0; i < N * N; i++){
		c[i] = a[i] - b[i];
	}
}

void addMats1D(float* a, float* b, float* c){
	for(int i = 0; i < N; i++){
		c[i] = a[i] + b[i];
	}
}

void subMats1D(float* a, float* b, float* c){
	for(int i = 0; i < N; i++){
		c[i] = a[i] - b[i];
	}
}

float* jacobiMethod(float* a, float* b, float* x){
	/*
	Mat2D* dinv = initMat2D(false);
	Mat2D* l = initMat2D(false);
	Mat2D* u = initMat2D(false);
	dluDecomp(a, dinv, l, u);
	int i = 0;
	while(i < 25){
		x = jacobiIterate(dinv, l, u, b, x);
		i++;
	}
	*/
	return x;
}

float* jacobiIterate(float* dinv, float* l, float* u, float* b, float* x){
	/*
	Mat2D* lu = addMats2D(l, u);
	Mat1D* lux = multiplyMats2D1D(lu, x);
	Mat1D* blux = subMats1D(b, lux);
	x = multiplyMats2D1D(dinv, blux);
	*/
	return x;
}

void dluDecomp(float* a, float* dinv, float* l, float* u){
	for(int i = 0; i < N; i++){
		for(int j = 0; j < N; j++){
			if(i == j){
				dinv[(i * N) + j] = 1 / a[(i * N) + j];
			}
			else if(i > j){
				dinv[(i * N) + j] = 0;
				l[(i * N) + j] = a[(i * N) + j];
			}
			else{
				dinv[(i * N) + j] = 0;
				u[(i * N) + j] = a[(i * N) + j];
			}
		}
	}
}
