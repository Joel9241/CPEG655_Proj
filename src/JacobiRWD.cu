#include "hip/hip_runtime.h"
#include "JacobiRWD.cuh"

float* initMat2D(bool init, bool host){
	return initMat2DHelper(init, host, N);
}

float* initMat2DHelper(bool init, bool host, int size){
	float* m;
	size_t sizeMat = sizeof(float) * size * size;
	if(host){
		m = (float*) malloc(sizeMat);
	}
	else{
		hipMalloc((void **) &m, sizeMat);
	}
	if(!init){
		return m;
	}
	for(int i = 0; i < size; i++){
		for(int j = 0; j < size; j++){
			m[(i * size) + j] = 5;
		}
	}
	return m;
}

float* initMat1D(bool init, bool host){
	return initMat1DHelper(init, host, N);
}

float* initMat1DHelper(bool init, bool host, int size){
	float* m;
	size_t sizeMat = sizeof(float) * size;
	if(host){
		m = (float*) malloc(sizeMat);
	}
	else{
		hipMalloc((void **) &m, sizeMat);
	}
	if(!init){
		return m;
	}
	for(int i = 0; i < size; i++){
		m[i] = 5;
	}
	return m;
}

void printMat1D(float* mat){
	printMat1DHelper(mat, N);
}

void printMat1DHelper(float* mat, int size){
	for(int i = 0; i < size; i++){
		printf("%f\n", mat[i]);
	}
}

void printMat2D(float* mat){
	printMat2DHelper(mat, N);
}

void printMat2DHelper(float* mat, int size){
	for(int i = 0; i < size; i++){
		for(int j = 0; j < size; j++){
			printf("%f ", mat[(i * size) + j]);
		}
		printf("\n");
	}
}

__global__ void multiplyMats2D(float* a, float* b, float* c){
	multiplyMats2DHelper(a, b, c, N, NT, NB);
}

__global__ void multiplyMats2DTB(float* a, float* b, float* c, int lN, int lNT, int lNB){
	multiplyMats2DHelper(a, b, c, lN, lNT, lNB);
}

__device__ void multiplyMats2DHelper(float* a, float* b, float* c, int lN, int lNT, int lNB){
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int bx = blockIdx.x;
	int by = blockIdx.y;
	for(int i = 0; i < lNB; i++){
		for(int j = 0; j < lNB; j++){
			float sum = 0;
			for(int k = 0; k < lN; k++){
				float tmp1 = a[(by * lN * lNT * lNB) + (ty * lN * lNB) + (j * lN) + k];
				float tmp2 = b[(bx * lNT * lNB) + (tx * lNB) + (k * lN) + i];
				sum += tmp1 * tmp2;
			}
			c[(by * lN * lNT * lNB) + (bx * lNT * lNB) + (ty * lN * lNB) + (j * lN) + i + (tx * lNB)] = sum;
		}
	}
}
__global__ void multiplyMats2D1D(float* a, float* b, float* c){
	multiplyMats2D1DHelper(a, b, c, N, NT, NB);
}

__global__ void multiplyMats2D1DTB(float* a, float* b, float* c, int lN, int lNT, int lNB){
	multiplyMats2D1DHelper(a, b, c, lN, lNT, lNB);
}

__device__ void multiplyMats2D1DHelper(float* a, float* b, float* c, int lN, int lNT, int lNB){
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int bx = blockIdx.x;
	int by = blockIdx.y;
	for(int i = 0; i < lNB; i++){
		float sum = 0;
		for(int k = 0; k < lN; k++){
			float tmp1 = a[(by * lN * lNT * lNB) + (tx * lN * lNB) + k];
			float tmp2 = b[(by * lN * lNT * lNB) + (ty * lN * lNB) + k];
			sum += tmp1 * tmp2;
		}
		c[(by * lN * lNT * lNB) + (bx * lNT * lNB) + (ty * lN * lNB) + i + (tx * lNB)] = sum;
	}
}

__global__ void addMats2D(float* a, float* b, float* c){
	addMatsHelper(a, b, c, N * N, NT, NB);
}

__global__ void addMats2DTB(float* a, float* b, float* c, int lN, int lNT, int lNB){
	addMatsHelper(a, b, c, lN * lN, lNT, lNB);
}

__global__ void addMats1D(float* a, float* b, float* c){
	addMatsHelper(a, b, c, N, NT, NB);
}

__global__ void addMats1DTB(float* a, float* b, float* c, int lN, int lNT, int lNB){
	addMatsHelper(a, b, c, lN, lNT, lNB);
}

__device__ void addMatsHelper(float* a, float* b, float* c, int lN, int lNT, int lNB){
	for(int i = 0; i < lN; i++){
		c[i] = a[i] + b[i];
	}
}

__global__ void subMats2D(float* a, float* b, float* c){
	subMatsHelper(a, b, c, N * N, NT, NB);
}

__global__ void subMats2DTB(float* a, float* b, float* c, int lN, int lNT, int lNB){
	subMatsHelper(a, b, c, lN * lN, lNT, lNB);
}

__global__ void subMats1D(float* a, float* b, float* c){
	subMatsHelper(a, b, c, N, NT, NB);
}

__global__ void subMats1DTB(float* a, float* b, float* c, int lN, int lNT, int lNB){
	subMatsHelper(a, b, c, lN, lNT, lNB);
}

__device__ void subMatsHelper(float* a, float* b, float* c, int lN, int lNT, int lNB){
	for(int i = 0; i < lN; i++){
		c[i] = a[i] - b[i];
	}
}

__global__ void jacobiMethod(float* a, float* b, float* x, float* dinv, float* l, float* u){
	/*
	int i = 0;
	while(i < 25){
		float* lu = initMat2DHelper(false, false, N);
		addMatsHelper(l, u, lu, N * N, NT, NB);
		float* lux = initMat1DHelper(false, false, N);
		multiplyMats2D1DHelper(lu, x, lux, N, NT, NB);
		float* blux = initMat1DHelper(false, false, N);
		subMatsHelper(b, lux, blux, N, NT, NB);
		multiplyMats2D1DHelper(dinv, blux, x, N, NT, NB);
		hipDeviceSynchronize();
		i++;
	}
	*/
}

__host__ void jacobiMethodTB(float* h_a, float* h_b, float* h_x, int lN, int lNT, int lNB, int lNK){
	dim3 threadPerBlock(lNT, lNT);
	dim3 blockPerGrid(lNK, lNK);

	float* h_dinv = initMat2DHelper(false, true, lN);
	float* h_l = initMat2DHelper(false, true, lN);
	float* h_u = initMat2DHelper(false, true, lN);
	float* h_lu = initMat2DHelper(false, true, lN);
	float* h_lux = initMat1DHelper(false, true, lN);
	float* h_blux = initMat1DHelper(false, true, lN);
																 
	float* d_a = initMat2DHelper(false, false, lN);
	float* d_b = initMat1DHelper(false, false, lN);
	float* d_x = initMat1DHelper(false, false, lN);
	float* d_dinv = initMat2DHelper(false, false, lN);
	float* d_l = initMat2DHelper(false, false, lN);
	float* d_u = initMat2DHelper(false, false, lN);
	float* d_lu = initMat2DHelper(false, true, lN);
	float* d_lux = initMat1DHelper(false, true, lN);
	float* d_blux = initMat1DHelper(false, true, lN);
	
	dluDecompTB(h_a, h_dinv, h_l, h_u, lN, lNT, lNB);

	size_t size1 = lN * lN * sizeof(float);
	size_t size2 = lN * sizeof(float);

	hipMemcpy(d_a, h_a, size1, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, size2, hipMemcpyHostToDevice);
	hipMemcpy(d_x, h_x, size2, hipMemcpyHostToDevice);
	hipMemcpy(d_dinv, h_dinv, size1, hipMemcpyHostToDevice);
	hipMemcpy(d_l, h_l, size1, hipMemcpyHostToDevice);
	hipMemcpy(d_u, h_u, size1, hipMemcpyHostToDevice);
	hipMemcpy(d_lu, h_lu, size1, hipMemcpyHostToDevice);
	hipMemcpy(d_lux, h_lux, size2, hipMemcpyHostToDevice);
	hipMemcpy(d_blux, h_blux, size2, hipMemcpyHostToDevice);

	int i = 0;
	while(i < 25){
		//printf("hello world\n");
		addMats2DTB<<<blockPerGrid, threadPerBlock>>>(d_l, d_u, d_lu, lN, lNT, lNB);
		hipDeviceSynchronize();
		multiplyMats2D1DTB<<<blockPerGrid, threadPerBlock>>>(d_lu, d_x, d_lux, lN, lNT, lNB);
		hipDeviceSynchronize();
		subMats1DTB<<<blockPerGrid, threadPerBlock>>>(d_b, d_lux, d_blux, lN, lNT, lNB);
		hipDeviceSynchronize();
		multiplyMats2D1DTB<<<blockPerGrid, threadPerBlock>>>(d_dinv, d_blux, d_x, lN, lNT, lNB);
		hipDeviceSynchronize();
		i++;
	}
	hipMemcpy(h_x, d_x, size2, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
}

void dluDecomp(float* a, float* dinv, float* l, float* u){
	for(int i = 0; i < N; i++){
		for(int j = 0; j < N; j++){
			if(i == j){
				dinv[(i * N) + j] = 1 / a[(i * N) + j];
			}
			else if(i > j){
				dinv[(i * N) + j] = 0;
				l[(i * N) + j] = a[(i * N) + j];
			}
			else{
				dinv[(i * N) + j] = 0;
				u[(i * N) + j] = a[(i * N) + j];
			}
		}
	}
}

void dluDecompTB(float* a, float* dinv, float* l, float* u, int lN, int lNT, int lNB){
	for(int i = 0; i < lN; i++){
		for(int j = 0; j < lN; j++){
			if(i == j){
				dinv[(i * lN) + j] = 1 / a[(i * lN) + j];
				l[(i * lN) + j] = 0.0;
				u[(i * lN) + j] = 0.0;
			}
			else if(i > j){
				dinv[(i * lN) + j] = 0.0;
				l[(i * lN) + j] = a[(i * lN) + j];
				u[(i * lN) + j] = 0.0;
			}
			else{
				dinv[(i * lN) + j] = 0.0;
				l[(i * lN) + j] = 0.0;
				u[(i * lN) + j] = a[(i * lN) + j];
			}
		}
	}
}
